//-*-C++-*-

#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

//#define TILE_WIDTH 10
#define TILE_DIM 8

__global__ void coalescedMultiply(float *a, float* b, float *c, int N)
{
  __shared__ float aTile[TILE_DIM][TILE_DIM];
  
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float sum = 0.0f;
  
  for (int p = 0; p < N/TILE_DIM; p++)
    {
      aTile[threadIdx.y][threadIdx.x] = a[row*N + (p*TILE_DIM + threadIdx.x)];
      __syncthreads();
      for (int i = 0; i < TILE_DIM; i++)
	{
	  sum += aTile[threadIdx.y][i]* b[i*N+col];
	}
      __syncthreads();
    }
  c[row*N+col] = sum;
}


int main()
{  
  cout << "Programme assumes that matrix size is N*N "<<endl;
  cout << "Please enter the N size number "<< endl;
  int N=0;
  cin >> N;

  // Initialize the memory on the host
  float *a, *b, *c;       
  
  // Initialize the memory on the device
  float *d_a, *d_b, *d_c; 
  
  // Allocate host memory
  a   = (float*)malloc(sizeof(float) * (N*N));
  b   = (float*)malloc(sizeof(float) * (N*N));
  c   = (float*)malloc(sizeof(float) * (N*N));
  
  // Initialize host arrays
  for(int i = 0; i < (N*N); i++)
    {
      a[i] = 2.0f;
      b[i] = 2.0f;
      c[i] = 2.0f;
    }
  
  // Allocate device memory
  hipMalloc((void**)&d_a, sizeof(float) * (N*N));
  hipMalloc((void**)&d_b, sizeof(float) * (N*N));
  hipMalloc((void**)&d_c, sizeof(float) * (N*N));
  
  // Transfer data from host to device memory
  hipMemcpy(d_a, a, sizeof(float) * (N*N), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * (N*N), hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, sizeof(float) * (N*N), hipMemcpyHostToDevice);
  
  // Thread organization
  int blockSize = 32;
  dim3 dimBlock(blockSize,blockSize,1);
  dim3 dimGrid(ceil(N/float(blockSize)),ceil(N/float(blockSize)),1);
  
  //  cout << dimBlock(blockSize,blockSize,1) << endl;
  //  cout << dimGrid(ceil(N/float(blockSize)),ceil(N/float(blockSize)),1) << endl;
  
  // Device fuction call 
  //  matrix_mul<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, N);
  coalescedMultiply<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, N);
  // Transfer data back to host memory
  hipMemcpy(c, d_c, sizeof(float) * (N*N), hipMemcpyDeviceToHost);

  // Verification
  for(int i = 0; i < N; i++)
    {
      for(int j = 0; j < N; j++)
      	{
	  cout << c[j*N+i] <<" ";
	}
      cout << " " <<endl;
    }
  
  // Deallocate device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  
  // Deallocate host memory
  free(a); 
  free(b); 
  free(c);
  
  return 0;
}
