//-*-C++-*-

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <time.h>

#define N 256
#define MAX_ERR 1e-6


// GPU function that adds two vectors 
__global__ void vector_add(float *a, float *b, 
	   float *out, int n) 
{
	
  int i	= blockIdx.x * blockDim.x * blockDim.y + 
    threadIdx.y * blockDim.x + threadIdx.x;   
  // Allow the   threads only within the size of N
  if(i < n)
    {
      out[i] = a[i] + b[i];
    }

  // Synchronice all the threads 
  __syncthreads();
}

int main()
{
  /*
  // Initialize the memory on the host
  float *a, *b, *out;

  // Allocate host memory
  a   = (float*)malloc(sizeof(float) * N);
  b   = (float*)malloc(sizeof(float) * N);
  out = (float*)malloc(sizeof(float) * N);
  */
   
  // Initialize the memory on the device
  float *d_a, *d_b, *d_out;

  // Allocate device memory
  hipMallocManaged(&d_a, sizeof(float) * N);
  hipMallocManaged(&d_b, sizeof(float) * N);
  hipMallocManaged(&d_out, sizeof(float) * N); 
  
  // Initialize host arrays
  for(int i = 0; i < N; i++)
    {
      d_a[i] = 1.0f;
      d_b[i] = 2.0f;
    }

  /*
  // Transfer data from host to device memory
  cudaMemcpy(d_a, a, sizeof(float) * N, cudaMemcpyHostToDevice);
  cudaMemcpy(d_b, b, sizeof(float) * N, cudaMemcpyHostToDevice);
  */

  // Thread organization 
  dim3 dimGrid(1, 1, 1);    
  dim3 dimBlock(16, 16, 1); 

  // execute the CUDA kernel function 
  vector_add<<<dimGrid, dimBlock>>>(d_a, d_b, d_out, N);
  hipDeviceSynchronize();
  /*
  // Transfer data back to host memory
  cudaMemcpy(out, d_out, sizeof(float) * N, cudaMemcpyDeviceToHost);
  */
  
  // Verification
  for(int i = 0; i < N; i++)
    {
      assert(fabs(d_out[i] - d_a[i] - d_b[i]) < MAX_ERR);
    }

  printf("out[0] = %f\n", d_out[0]);
  printf("PASSED\n");
    
  // Deallocate device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);

  /*
  // Deallocate host memory
  free(a); 
  free(b); 
  free(out);
  */

  return 0;
}
